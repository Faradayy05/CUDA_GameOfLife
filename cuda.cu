#include "hip/hip_runtime.h"
%%cu
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <string>
#include <sstream>
#include <cstring>
#include <chrono>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU assert: %s %s line %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void printArrAsField(int* field, int fieldSize) {
  for (int i = 0; i < fieldSize; i++) {
      for (int j = 0; j < fieldSize; j++) {
        if (field[i*fieldSize + j] == 0) {
          cout << '.';
        } else {
          cout << 'O';
        }
      }
    cout << endl;
  }
  cout << endl;
}

void printRaw(int* field, int fieldSize) {
  for (int i = 0; i < fieldSize; i++) {
      for (int j = 0; j < fieldSize; j++) {
        cout << field[i*fieldSize + j] << '\t';
      }
    cout << endl;
  }
  cout << endl;
}


__global__ void evalCell(int *prevField, int *newField, int fieldSize) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx < fieldSize*fieldSize) {
    const int around[8][2] = {
      {-1, 0}, // Left
      {1, 0},  // Right
      {0, -1}, // Up
      {0, 1},  // Down
      {-1, -1}, // Up Left
      {-1, 1},  // Down Left
      {1, -1}, // Up Right
      {1, 1},  // Up Down
    };

    // Count living neighbour
    int count = 0;
    for (int i = 0; i < 8; i++) {
      int aroundIdx = idx + around[i][0] + around[i][1] * fieldSize;
      if (aroundIdx > 0 && aroundIdx < fieldSize * fieldSize && (aroundIdx+1) % fieldSize != 0 && (aroundIdx-1) % fieldSize != 0) {
        if (prevField[aroundIdx] == 1) {
          count++;
        }
      }		
    }

    // Evaluate cell
    if (prevField[idx] == 1 && count != 2 && count != 3) {
      newField[idx] = 0;				
    } else if (newField[idx] == 0 && count == 3) {
      newField[idx] = 1;
    } else {
      newField[idx] = prevField[idx];
    }
  }
}

void evalField(int *field, int fieldSize) {
  // Declare grid size and block size
  const int BLOCK_SIZE = 512;
  const int GRID_SIZE = fieldSize * fieldSize / THREAD_SIZE + 1;

  // Declare GPU memory pointers
  size_t BYTES = fieldSize * fieldSize * sizeof(int);
  int *d_in, *d_out;

  // Allocate GPU memory
  gpuErrchk(hipMalloc((void**) &d_in, BYTES));
  gpuErrchk(hipMalloc((void**) &d_out, BYTES));

  // Transfer the array to the GPU
  gpuErrchk(hipMemcpy(d_in, field, BYTES, hipMemcpyHostToDevice));

  evalCell<<<GRID_SIZE, BLOCK_SIZE>>>(d_in, d_out, fieldSize);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk(hipMemcpy(field, d_out, BYTES, hipMemcpyDeviceToHost));

  hipFree(d_in);
  hipFree(d_out);
}

void initField(int *field, int fieldSize) {
  // Initialize field with pattern using file
  // File format:
  // x1 y1
  // x2 y2
  // x3 y3
  // etc
  
  // Cell (x, y) will become living cell
  ifstream fin("field-gosper.txt");

  if (fin) {
    string lineread;
    while (getline(fin, lineread)) {
        stringstream line(lineread);
        string coor;
        line >> coor;
        int x = stoi(coor);
        line >> coor;
        int y = stoi(coor);
        
        // // One
        // field[y][x] = 1;
        
        // Repeating pattern
        const int TEMPLATE_SIZE = 40; // Max size of pattern generated
        for (int i = 0; i < fieldSize / TEMPLATE_SIZE; i++) {  
          for (int j = 0; j < fieldSize / TEMPLATE_SIZE; j++) {  
            field[i*TEMPLATE_SIZE*fieldSize + y*fieldSize + j*TEMPLATE_SIZE+ x] = 1;
          }
        }
    }
    fin.close();
  }
}

int main() {
  int iter = 1000;
  int n;
  n = 512;
  
  int *field;
  field = (int*) malloc(sizeof(int) * n*n);
  
  initField(field, n);

  ofstream fout;
  fout.open("output-cuda.txt", ofstream::out | ofstream::trunc);

  auto start = chrono::steady_clock::now();

  for (int i = 0; i < iter; i++) {
	evalField(field, n);

    //output the field to file
    for (int i = 0; i < n; i++) {
     for (int j = 0; j < n; j++) {
        if (field[i*n + j] == 0) {
          fout << '.';
        } else {
          fout << 'O';
        }
      }
      fout << endl;
    }
    fout << endl;
	}

  auto end = chrono::steady_clock::now();

  auto diff = end - start;
  cout << "CUDA:" << endl;
  cout << "Field size: " << n << endl;
  cout << "Iteration: " << iter << endl;
  cout << "Time: " << chrono::duration <double, milli> (diff).count() / 1000 << " seconds" << endl;

  fout.close();
	
  free(field);
	
  return 0;
}